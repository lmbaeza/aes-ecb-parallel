#include "hip/hip_runtime.h"
%%writefile cuda_sample.cu
#include <stdio.h>

#include<vector>
#include<iostream>
#include<string>
#include<cstdint>
#include<cstring>
#include<cassert>
#include<fstream>

#include <hip/hip_runtime.h>

#include "aes_cuda.h"
#include "file_handler_cuda.h"

using namespace std;

string NOMBRE_ARCHIVO;
string ARCHIVO_SALIDA;

int NUM_HILOS;

int *pArgc = NULL;

char** pArgv = NULL;

// el i-th hilo procesa el intervalo, [from, to]
int (*intervalo)[2];

// texto dividido en bloques: la i-th posicion tiene el i-th bloque
uint8 (*text_hex)[BLOCKS_SIZE] = NULL;

// bloque de 32 posiciones que representa la llave
uint8* key_hex = NULL;

// Mensaje encriptado: la i-th posicion tiene el i-th bloque
uint8 (*cipher_text)[BLOCKS_SIZE] = NULL;

// Instancia global del AES

int blocks;

// __device__
// int d_blocks;

void build_hex(string &text, string &key) {
    int n = (int) text.size();
    blocks = (n+BLOCKS_SIZE-1)/BLOCKS_SIZE;

    text_hex = new uint8[blocks][BLOCKS_SIZE];
    // for(int i = 0; i < blocks; ++i)
    //     text_hex[i] = new uint8[BLOCKS_SIZE];

    text_to_hex(text, text_hex);

    key_hex = (uint8 *) malloc(32*sizeof(uint8));
    for(int i = 0; i < 32; ++i) key_hex[i] = 0;
    key_to_hex(key, key_hex);
}

void build_ranges(int text_size) {
    intervalo = new int[NUM_HILOS][2];
    for(int i = 0; i < NUM_HILOS; ++i) {
        // intervalo[i] = new int[2];
        intervalo[i][0] = 0;
        intervalo[i][1] = 0;
    }
    
    // for(int i = 0; i < NUM_HILOS; ++i) {
    //     intervalo[i] = (int *) malloc(2*sizeof(int));
    //     intervalo[i][0] = 0;
    //     intervalo[i][1] = 0;
    // }

    blocks = (text_size+32-1)/32;
    
    cipher_text = new uint8[blocks][BLOCKS_SIZE];

    // Crear Intervalos
    int len = blocks/NUM_HILOS;
    vector<int> length(NUM_HILOS);
    for(int i = 0; i < NUM_HILOS; ++i) length[i] = len;
    int total = blocks - len*NUM_HILOS;
    for(int i = 0; i < total; ++i) length[i]++;
    int current = 0;
    for(int i = 0; i < NUM_HILOS; ++i) {
        intervalo[i][0] = current;
        intervalo[i][1] = current+length[i]-1;
        current = current + length[i];
    }
    intervalo[NUM_HILOS-1][1] = blocks-1;
}


__global__ 
void kernel(int (*k_intervalo)[2], uint8 (*k_cipher_text)[BLOCKS_SIZE], uint8 (*k_text_hex)[BLOCKS_SIZE], uint8* k_key_hex, AES* aes, int * d_blocks) {
    int ID = blockIdx.x * blockDim.x + threadIdx.x;

    printf("\nGPU ID=%d blocks=%d", ID, (*d_blocks));

    if (ID < (*d_blocks)) {

        printf(" -- Inside ID=%d\n", ID);
        int from = k_intervalo[ID][0];
        int to = k_intervalo[ID][1];
        
        printf("\nGPU from=%d to=%d\n", from, to);

        for(int i = from; i <= to; ++i) {
            printf("GPU #1\n");
            uint len = 0;
            printf("GPU #2\n");
            uint8* cipher = aes->EncryptECB(k_text_hex[i], 16 * sizeof(uint8), k_key_hex, len);
            printf("GPU #3\n");
            for(int j = 0; j < BLOCKS_SIZE; ++j) {
                k_cipher_text[i][j] = cipher[j];
            }
            //k_cipher_text[i] = cipher;

            printf("GPU: ");
            for(int j = 0; j < 16; ++j) {
                uint8 val = k_cipher_text[i][j];
                printf("%02x ", val);
            }
            printf("\n");
        }
        // k_cipher_text vá a ser el unico modificado
    }
}

int main(int argc, char **argv) {   
    pArgc = &argc;
    pArgv = argv;
 
    if((*pArgc) < 4) {
        printf("Debe proporcionar 3 argumentos: [archivo de entrada] [archivo de salida] [numero de hilos]");
        // Ejemplo: ./filtro.o img/input1.png img/output1.png 8 16
        exit(0);
    }
    // Ruta del archivo de entrada: Ej: input1.txt
    NOMBRE_ARCHIVO = string(pArgv[1]);

    // Ruta del texto cifrado: Ej: output.bin
    ARCHIVO_SALIDA = string(pArgv[2]);

    // Numero de hilos utilizados
    NUM_HILOS = atoi(pArgv[3]);
 
    printf("%s %s %d\n", NOMBRE_ARCHIVO.c_str(), ARCHIVO_SALIDA.c_str(), NUM_HILOS);

    string text;
    string key = "admin1234"; // maximo 32 caracteres

    // Leer el texto que se vá a encriptar
    read_file_to_string(NOMBRE_ARCHIVO, text);

    text = "12345343243534534564365656546546";
    printf("Text: [%s]\n", text.c_str());
    
    build_hex(text, key);

    // Crear los rangos donde van a trabajar lso
    build_ranges((int) text.size());


    int (*d_intervalo)[2];
    hipMalloc(&d_intervalo, NUM_HILOS*2*sizeof(int));
    hipMemcpy(d_intervalo, intervalo, NUM_HILOS*2*sizeof(int), hipMemcpyHostToDevice);

    // cipher_text
    uint8 (*d_cipher_text)[BLOCKS_SIZE];
    hipMalloc(&d_cipher_text, blocks*BLOCKS_SIZE*sizeof(uint8));
    hipMemcpy(d_cipher_text, cipher_text,  blocks*BLOCKS_SIZE*sizeof(uint8), hipMemcpyHostToDevice);

    // text_hex
    uint8 (*d_text_hex)[BLOCKS_SIZE];
    // float (*d_C)[N];
    hipMalloc((void**)(&d_text_hex), blocks*BLOCKS_SIZE*sizeof(uint8));

    hipMemcpy(d_text_hex, text_hex,  blocks*BLOCKS_SIZE*sizeof(uint8), hipMemcpyHostToDevice);

    // key_hex
    uint8* d_key_hex = NULL;
    hipMalloc(&d_key_hex, 32*sizeof(uint8));
    hipMemcpy(d_key_hex, key_hex,  32*sizeof(uint8), hipMemcpyHostToDevice);

    AES aes(256);
    AES *d_aes;
    hipMalloc(&d_aes, sizeof(AES));
    hipMemcpy(d_aes, &aes,  sizeof(AES), hipMemcpyHostToDevice);

    printf("CPU blocks=%d\n", blocks);

    int *d_blocks;
    hipMalloc(&d_blocks, sizeof(int));

    hipMemcpy(d_blocks, &blocks,  sizeof(int), hipMemcpyHostToDevice);
    
    kernel<<<1, 1>>>(d_intervalo, d_cipher_text, d_text_hex, d_key_hex, d_aes, d_blocks);
    hipDeviceSynchronize();

    hipMemcpy(cipher_text, d_cipher_text, blocks*BLOCKS_SIZE*sizeof(uint8), hipMemcpyDeviceToHost);

    // write_file(ARCHIVO_SALIDA, cipher_text, blocks, 16);

    printf("###################################\n");

    for(int i = 0; i < blocks; ++i) {
        for(int j = 0; j < 16; ++j) {
            uint8 val = cipher_text[i][j];
            if(val < 16) cout << '0';
            cout << hex << (int) val;
        }
        cout << '\n';
    }

    return 0;
}

// 78cdb9aa782851e8502e5d8da6927b25